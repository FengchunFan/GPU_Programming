//CUDA version of vector_add program
//program computes the addition of two vectors stored in array a and b, and put the result in out.

//OBSERVATION: 
//with only __global__ specifier and function call, the program will execute but the functionality will not be performed
//we can see the first index of the out still being 0 instead of designated 3
//The reason is that we need to allocate device (GPU) memory manually and perform data transfer between host (CPU) and device memory
//Following steps in the note
//time: real 0m1.063s


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>

#define N 10000000

//add __global__ specifier for the function to execute on GPU
__global__ void vector_add(float *out, float *a, float *b, int n) { //GPU kernel
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; //CPU
    float *d_a, *d_b, *d_out; //GPU

    // Allocate memory for host
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Allocate memory for device
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_out, out, sizeof(float) * N, hipMemcpyHostToDevice);

    // Main function
    //vector_add(out, a, b, N);

    //need to change function call to CUDA's way of function call
    vector_add<<<1,1>>>(d_out, d_a, d_b, N);

    //need to transfer data back from 
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    printf("first index first: %f\n", out[0]);
    printf("last index last: %f\n", out[9999999]);
    printf("out of boundary index: %f\n", out[10000000]);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    free(a);
    free(b);
    free(out);
}
