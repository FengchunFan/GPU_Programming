//CUDA version of vector_add program
//program computes the addition of two vectors stored in array a and b, and put the result in out.
//running parallelly with mulitple threads and 1 thread block on GPU

//OBSERVATION: 
//with only __global__ specifier and function call, the program will execute but the functionality will not be performed
//we can see the first index of the out still being 0 instead of designated 3
//The reason is that we need to allocate device (GPU) memory manually and perform data transfer between host (CPU) and device memory
//Following steps in the note

#include <stdio.h>
#include <cstdlib>
#include <hip/hip_runtime.h>


#define N 10000000

//add __global__ specifier for the function to execute on GPU
__global__ void vector_add(float *out, float *a, float *b, int n) {
    int index = threadIdx.x; //range from 0 to 255
    int stride = blockDim.x; //constant 256

    for(int i = index; i < n; i += stride){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; //CPU
    float *d_a, *d_b, *d_out; //GPU

    // Allocate memory for host
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Allocate memory for device
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    //need to change function call to CUDA's way of function call
    vector_add<<<1,256>>>(d_out, d_a, d_b, N); //running with 1 thread block and 256 threads

    //need to transfer data back from 
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    printf("first index first: %f\n", out[0]);
    printf("last index last: %f\n", out[9999999]);
    printf("out of boundary index: %f\n", out[10000000]);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    free(a);
    free(b);
    free(out);
}
