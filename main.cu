//sample Hello world CUDA program

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello() {
    printf("Hello World from GPU!\n");
}

int main() {
    cuda_hello<<<1, 1>>>();
    hipDeviceSynchronize();  // Wait for GPU kernel to complete
    return 0;
}