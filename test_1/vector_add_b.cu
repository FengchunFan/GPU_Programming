//CUDA version of vector_add program
//program computes the addition of two vectors stored in array a and b, and put the result in out.
//running parallelly with mulitple threads and multiple thread blocks on GPU

//OBSERVATION: 
//with only __global__ specifier and function call, the program will execute but the functionality will not be performed
//we can see the first index of the out still being 0 instead of designated 3
//The reason is that we need to allocate device (GPU) memory manually and perform data transfer between host (CPU) and device memory
//Following steps in the note

#include <stdio.h>
#include <cstdlib>
#include <hip/hip_runtime.h>


#define N 10000000

//add __global__ specifier for the function to execute on GPU
__global__ void vector_add(float *out, float *a, float *b, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n){
        out[tid] = a[tid] + b[tid];
    }
}

int main(){
    float *a, *b, *out; //CPU
    float *d_a, *d_b, *d_out; //GPU

    // Allocate memory for host
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Allocate memory for device
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Main function
    //vector_add(out, a, b, N);

    //need to change function call to CUDA's way of function call
    int block_size = 256;
    int grid_size = ((N + block_size) / block_size);
    vector_add<<<grid_size,block_size>>>(d_out, d_a, d_b, N);

    //need to transfer data back from 
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    printf("first index first: %f\n", out[0]);
    printf("last index last: %f\n", out[9999999]);
    printf("out of boundary index: %f\n", out[10000000]);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    free(a);
    free(b);
    free(out);
}
